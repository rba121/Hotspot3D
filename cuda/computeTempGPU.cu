#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../common/my_timer.h"

#define MAX_PD	(3.0e6)
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
#define FACTOR_CHIP	0.5
#define AMB_TEMP 80.0

__global__ void computeTemp(float *p, float* tIn, float *tOut, float sdc,
        int nx, int ny, int nz,
        float ce, float cw, 
        float cn, float cs,
        float ct, float cb, 
        float cc) 
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;  
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    float bottom, center, top;
    bottom = center = tIn[c];
    top = tIn[c+xy];
    tOut[c] = cc * center + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * bottom + ct * top + sdc * p[c] + ct * AMB_TEMP;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        bottom = center;
        center = top;
        top = tIn[c+xy];
        tOut[c] = cc * center + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
            + cn * tIn[N] + cb * bottom + ct * top + sdc * p[c] + ct * AMB_TEMP;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    bottom = center;
    center = top;
    tOut[c] = cc * center + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * bottom + ct * top + sdc * p[c] + ct * AMB_TEMP;
    return;
}

void computeTempGPU(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter) 
{
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(float) * nx * ny * nz;  
    float  *tIn_d, *tOut_d, *p_d;

	hipStream_t stream;
	hipStreamCreate(&stream);

	timespec timer = tic();

    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpyAsync(tIn_d, tIn, s, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(p_d, p, s, hipMemcpyHostToDevice, stream);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(computeTemp), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);

    for (int i = 0; i < numiter; ++i) {
        computeTemp<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);

        float *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }

	hipMemcpyAsync(tOut, tOut_d, s, hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	toc(&timer, "GPU execution time (including data transfer) in seconds");

    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}

void initializeArrays(float* pIn, float* tIn, const int nx, const int ny, const int nz)
{
	int z,x,y;	

	for (z = 0; z < nz; z++) {
		for (x = 0; x < nx; x++) {
			for (y = 0; y < ny; y++) {
				pIn[z * nx * ny + x * ny + y] = (float)((z * x * y + 1) % nz) / nz;
			}
		}
	}

	for (z = 0; z < nz; z++) {
		for (x = 0; x < nx; x++) {
			for (y = 0; y < ny; y++) {
				tIn[z * nx * ny + x * ny + y] = (float)((z * x * y + 1) % nx) / nx;
			}
		}
	}
}

float accuracy(float *arr1, float *arr2, int len)
{
    float err = 0.0; 
    int i;
    for(i = 0; i < len; i++)
    {
        err += abs(arr1[i] - arr2[i]);
    }

    return (float)err / len;
}

void computeTempCPU(float *pIn, float* tIn, float *tOut, 
        int nx, int ny, int nz, float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter) 
{   float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    int c,w,e,n,s,b,t;
    int x,y,z;
    int i = 0;
    do{
        for(z = 0; z < nz; z++)
            for(y = 0; y < ny; y++)
                for(x = 0; x < nx; x++)
                {
                    c = x + y * nx + z * nx * ny;

                    w = (x == 0) ? c      : c - 1;
                    e = (x == nx - 1) ? c : c + 1;
                    n = (y == 0) ? c      : c - nx;
                    s = (y == ny - 1) ? c : c + nx;
                    b = (z == 0) ? c      : c - nx * ny;
                    t = (z == nz - 1) ? c : c + nx * ny;


                    tOut[c] = tIn[c]*cc + tIn[n]*cn + tIn[s]*cs + tIn[e]*ce + tIn[w]*cw + tIn[t]*ct + tIn[b]*cb + (dt/Cap) * pIn[c] + ct*AMB_TEMP;
                }
        float *temp = tIn;
        tIn = tOut;
        tOut = temp; 
        i++;
    }
    while(i < numiter);
}

int main(int argc, char* argv[]) {
	if (argc != 4) {
		printf("Usage: ./computeTempFPGA <NX/NY> <NZ> <NUM_ITERATIONS>\n");
	}

	const int NX = strtol(argv[1], NULL, 10);
	const int NY = NX;
	const int NZ = strtol(argv[2], NULL, 10);
	const int NUM_ITERATIONS = strtol(argv[3], NULL, 10);

	const float t_chip = 0.0005;
	const float chip_height = 0.016;
	const float chip_width = 0.016; 
    const float dx = chip_height / NX;
    const float dy = chip_width / NY;
    const float dz = t_chip / NZ;
    const float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * dx * dy;
    const float Rx = dy / (2.0 * K_SI * t_chip * dx);
    const float Ry = dx / (2.0 * K_SI * t_chip * dy);
    const float Rz = dz / (K_SI * dx * dy);
    const float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    const float dt = PRECISION / max_slope;

    const int size = NX * NY * NZ;

    float* powerIn;
	hipHostAlloc(&powerIn, size * sizeof(float), hipHostMallocDefault);
    float* tempIn;
	hipHostAlloc(&tempIn, size * sizeof(float), hipHostMallocDefault);
    float* tempOut;
	hipHostAlloc(&tempOut, size * sizeof(float), hipHostMallocDefault);
    float* tempCopy = (float*)malloc(size * sizeof(float));
    float* answer = (float*)calloc(size, sizeof(float));

	initializeArrays(powerIn, tempIn, NX, NY, NZ);
    memcpy(tempCopy, tempIn, size * sizeof(float));

	computeTempGPU(powerIn, tempIn, tempOut, NX, NY, NZ, Cap, Rx, Ry, Rz, dt, NUM_ITERATIONS);
	hipHostFree(tempIn);
	computeTempCPU(powerIn, tempCopy, answer, NX, NY, NZ, Cap, Rx, Ry, Rz, dt, NUM_ITERATIONS);
	printf("Accuracy: %e\n", accuracy(tempOut, answer, size));

	hipHostFree(powerIn);
	hipHostFree(tempOut);
	free(tempCopy);
	free(answer);

	return 0;
}
